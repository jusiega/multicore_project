#include "hip/hip_runtime.h"
#ifndef _POSIX_C_SOURCE
#define _POSIX_C_SOURCE 199309L
#endif
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <unistd.h>


#define STB_IMAGE_IMPLEMENTATION
#include "../Common/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Common/stb_image_write.h"








double wtime()
{
static int sec = -1;
struct timespec tv;
clock_gettime(CLOCK_MONOTONIC, &tv);
if (sec < 0) sec = tv.tv_sec;
return (tv.tv_sec - sec) + 1.0e-9*tv.tv_nsec;
}



void fblur(float* r, float* g, float* b, float* r2, float* g2, float* b2, int width, int height, float*  blur, int blursize) //standard blur
{
    
    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;

    int shift=(blursize-1)/2;

    for(int i=shift; i<height-shift; i++)
    {
        for(int j=shift; j<width-shift; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(i-shift+k)*width + (j-shift+l)]);
                    green_tmp+=blur[k*blursize+l]*(g[(i-shift+k)*width + (j-shift+l)]);
                    blu_tmp+=blur[k*blursize+l]*(b[(i-shift+k)*width + (j-shift+l)]);
                    //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
                }
                
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //EDG TOP
    for(int i=0; i<shift; i++)
    {
        for(int j=0; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //EDG left
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<shift; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //EDGE DOWN/RIGHT
    for(int i=height-shift; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    for(int i=0; i<height; i++)
    {
        for(int j=width-shift; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }    


    for(int i=0; i<height; i++) 
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=r2[i*width+j];
            g[i*width+j]=g2[i*width+j];
            b[i*width+j]=b2[i*width+j];
        }
    }



}






void sepfilter(float* r, float* g, float* b, float* rnew, float* gnew, float* bnew, int width, int height, float*  blurh, float* blurv, int blursize) //blur function for spearable filters, much speed wow
{
    
    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;
    int shift=(blursize-1)/2;

    //MAIN BLUR VERTICAL 
    for(int i=shift; i<height-shift; i++)
    {
        for(int j=0; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {

                //red_tmp+=blurh[k]*float(r[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //green_tmp+=blurh[k]*float(g[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //blu_tmp+=blurh[k]*float(b[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
                red_tmp+=blurv[k]*(r[(i-shift)*width+k*width + j]);
                green_tmp+=blurv[k]*(g[(i-shift)*width+k*width + j]);
                blu_tmp+=blurv[k]*(b[(i-shift)*width+k*width + j]);

                
            }
            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }

    //EDGE UP USING MIRROR & VERTICAL BLUR
    //SOMEBODY ONCE TOLD ME THAT IF STATEMENTS ARE SLOW
    for(int i=0; i<shift; i++)
    {
        for(int j=0; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                red_tmp+=blurv[k]*(r[abs(((i-shift)+k))*width + j]);
                green_tmp+=blurv[k]*(g[abs(((i-shift)+k))*width + j]);
                blu_tmp+=blurv[k]*(b[abs(((i-shift)+k))*width + j]);
            }
            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //EDGE DOWN
    for(int i=height-shift; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            for (int k=0; k<blursize; k++)
            {
                red_tmp+=blurv[k]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
                green_tmp+=blurv[k]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
                blu_tmp+=blurv[k]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
            }
            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //MAIN BLUR HORIZONTAL, NO LEFT & RIGHT EDGES
    for(int i=0; i<height; i++)
    {
        for(int j=shift; j<width-shift; j++)
        {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + j-shift+l]);
                green_tmp+=blurh[l]*(gnew[i*width + j-shift+l]);
                blu_tmp+=blurh[l]*(bnew[i*width + j-shift+l]);
                //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
            }
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }

    //EDGE LEFT HORIZONTAL PART
    //VERTICAL PART WAS DONE IN MAIN BLUR VERTICAL
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<shift; j++)
        {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + abs((j-shift+l))]);
                green_tmp+=blurh[l]*(gnew[i*width + abs((j-shift+l))]);
                blu_tmp+=blurh[l]*(bnew[i*width + abs((j-shift+l))]);
            }
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }
    //EDG RIGHT
    for(int i=0; i<height; i++)
    {
        for(int j=width-shift; j<width; j++)
        {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + width-1-abs(width-1-(j-shift+l))]);
                green_tmp+=blurh[l]*(gnew[i*width + width-1-abs(width-1-(j-shift+l))]);
                blu_tmp+=blurh[l]*(bnew[i*width + width-1-abs(width-1-(j-shift+l))]);
            }
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
        }
    }




}



void clamp(float* r, float* g, float* b, int width, int height) /////CLAMP TO RGB RANGE
{
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            if (r[i*width+j]>255.0)
            {
                r[i*width+j]=255.0;
            } 
            else if (r[i*width+j]<0)
            {
                r[i*width+j]=0;
            }
            
            if (g[i*width+j]>255.0)
            {
                g[i*width+j]=255.0;
            }
            else if (g[i*width+j]<0)
            {
                g[i*width+j]=0;
            }
            
            
            if (b[i*width+j]>255.0)
            {
                b[i*width+j]=255.0;
            }
            else if (b[i*width+j]<0)
            {
                b[i*width+j]=0;
            }
            
            
        }
    }
}





int main()
{
    
    std::cout<<"welcome. input (separated by spaces) your source image path, your chosen destination path and desired operation and parameters for it."<<std::endl<<"0: gaussian blur (parameter: standard deviation (px, float))  ///  1: box blur (parameter: size (px, int, odd))"<<std::endl<<"2: circular box blur (bokeh) (parameter: diameter (px, int, odd))  ///  3: sobel edge detection (parameters: vertical detection, horizontal detection (bool))"<<std::endl<<"4: difference of gaussians edge detection (2 parameters: stdev1, stdev2 (float>0))  ///  5:identity transformation (parameter: size (int)) (useless)"<<std::endl<<"6: unsharp mask simple (parameter: strength (float))  ///  7: unseparated (slow) box blur (parameter: size(px, int, odd))"<<std::endl<<"always type in two numbers as parameters, if the process uses only one parameter set the second one to whatever. 0 or less sets parameter to its default value"<<std::endl;
    std::cout<<"example input: '../samples/default.bmp ../output/test.bmp 0 7 0'"<<std::endl<<std::endl;
    std::string source="default.bmp";
    std::string destination="output/test.bmp";
    int width=0;
    int height=0;
    int sw=0;
    float p1=0;
    float p2=0;
    int p3=0;
    int nchannels=3;
    std::cin>>source>>destination>>sw>>p1>>p2;
    double totaltime=0;
    double elapsedtime=wtime();
    double t0=elapsedtime;
    //check if image exists, is valid format and if yes then query values of width, height and channels
    if (stbi_info(source.c_str(), &width, &height, &nchannels)==0)
    {
        std::cout<<"Invalid source. Aborting mission.";
        return 0;
    }

    unsigned char *data=stbi_load(source.c_str(), &width, &height, &nchannels, 0);
    //std::cout<<wtime()<<std::endl;    


    float* r=(float*)malloc(width*height*sizeof(float));
    float* g=(float*)malloc(width*height*sizeof(float));
    float* b=(float*)malloc(width*height*sizeof(float));
    float* rnew=(float*)malloc(width*height*sizeof(float));
    float* gnew=(float*)malloc(width*height*sizeof(float));
    float* bnew=(float*)malloc(width*height*sizeof(float));
    

    //std::cout<<"aeeeeee"<<r[0]<<std::endl;

    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=float(data[i*width*nchannels+j*nchannels]);
            g[i*width+j]=float(data[i*width*nchannels+j*nchannels+1]);
            b[i*width+j]=float(data[i*width*nchannels+j*nchannels+2]);
            //std::cout<<int(data[i*width*nchannels+j*nchannels])<<int(data[i*width*nchannels+j*nchannels+1])<<int(data[i*width*nchannels+j*nchannels+2])<<std::endl;
        }
    }  

    //std::cout<<"aeeeeee"<<std::endl;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////IMYDŻ LOADED
 
    int blursize=3;//keep this odd
    float identity3[]={0,0,0,0,1,0,0,0,0};

switch (sw)
{
case 0:// gausjan
{    ////////GAUS BLUR/////////////    
    float stdev=2;//default
    if (p1>0)
    {
        stdev=p1;
    }
    int gaussize=(2*int(stdev)-1)*3;// 3 sigma wide blur
    if (int(stdev)==0)
    {
        gaussize=3;
    }
    float* gausblur=(float*)malloc(gaussize*sizeof(float));
    float norm=0;
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize-1)/2,2)/(2*stdev*stdev));
        norm+=gausblur[i];
    }
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    sepfilter(r,g,b,rnew,gnew,bnew,width,height,gausblur,gausblur,gaussize);
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed during filtering: "<<elapsedtime<<" s."<<std::endl;
    free(gausblur);
}
break;
    
    
case 1://box blur (separable)
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* boxblursep=(float*)malloc(blursize*sizeof(float));
    for(int i=0;i<blursize;i++)
    {
        boxblursep[i]=1.0/float(blursize);
        //std::cout<<boxblursep[i]<<", ";
    }
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl; 
    sepfilter(r,g,b,rnew,gnew,bnew,width,height,boxblursep,boxblursep,blursize);
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    free(boxblursep);
}
break;


  ///////BOKEH (CIRCULAR BOX BLUR) obviously one might say there are no perfect circles in any image composed of pixels and while this is one hundred percent true one may see that they cant actually see the uncircleness beyond a certain point which I shall now take the advantage of
case 2://bokeh very slow not separed
{
    int diameter=5;/////diameter in pixels keep this odd pls
    if (p1>0)
    {
        diameter=(int)(2*ceil(p1/2)-1);
    }
    int radius=(diameter-1)/2;
    float* bokeh=(float*)malloc(diameter*diameter*sizeof(float));
    float R=0;
    float sb=0;
    for(int i=0; i<diameter; i++)//initialise
    {
        for(int j=0; j<diameter; j++)
        {
            R=sqrt((j-radius)*(j-radius)+(i-radius)*(i-radius));
            if (R<=radius)
            {
                bokeh[i*diameter+j]=1;
                sb+=1;
            }
            else
            {
                bokeh[i*diameter+j]=0;
            }
        }
    }
    for(int i=0; i<diameter; i++)
    {
        for(int j=0; j<diameter; j++)
        {
            bokeh[i*diameter+j]/=(sb);
            //std::cout<<bokeh[i*diameter+j]<<" ";
        }
    }
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;  
    fblur(r,g,b,rnew,gnew,bnew,width,height,bokeh,diameter);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    free(bokeh);
}
break;

case 3://sobel
{
    bool ver=(bool)int(p1);
    bool hor=(bool)int(p2);
    float* r2=(float*)malloc(width*height*sizeof(float));
    float* g2=(float*)malloc(width*height*sizeof(float));
    float* b2=(float*)malloc(width*height*sizeof(float));
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r2[i*width+j]=r[i*width+j];
            g2[i*width+j]=g[i*width+j];
            b2[i*width+j]=b[i*width+j];
        }
    }  
    float gxv[]={1,2,1};
    float gxh[]={1,0,-1};
    float gyv[]={1,0,-1};
    float gyh[]={1,2,1};
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;   
    sepfilter(r,g,b,rnew,gnew,bnew,width,height,gxh,gxv,3);
    sepfilter(r2,g2,b2,rnew,gnew,bnew,width,height,gyh,gyv,3);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    
    if(ver && hor)
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=sqrt(r[i*width+j]*r[i*width+j]+r2[i*width+j]*r2[i*width+j]);
                g[i*width+j]=sqrt(g[i*width+j]*g[i*width+j]+g2[i*width+j]*g2[i*width+j]);
                b[i*width+j]=sqrt(b[i*width+j]*b[i*width+j]+b2[i*width+j]*b2[i*width+j]);
            }
        }  
    }
    else if (ver)
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=r2[i*width+j];
                g[i*width+j]=g2[i*width+j];
                b[i*width+j]=b2[i*width+j];
            }
        }
    }
    else if (hor)
    {
        
    }
    else
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=0;
                g[i*width+j]=0;
                b[i*width+j]=0;
            }
        }
    }
    free(r2);
    free(g2);
    free(b2);
}
break;


case 4: //TWO GAUS
{
    float* r2=(float*)malloc(width*height*sizeof(float));
    float* g2=(float*)malloc(width*height*sizeof(float));
    float* b2=(float*)malloc(width*height*sizeof(float));
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r2[i*width+j]=r[i*width+j];
            g2[i*width+j]=g[i*width+j];
            b2[i*width+j]=b[i*width+j];
        }
    }  
    float stdev=2;//default
    if (p1>0)
    {
        stdev=p1;
    }
    int gaussize=(2*int(stdev)-1)*3;// 3 sigma wide blur
    if (int(stdev)==0)
    {
        gaussize=3;
    }
    float* gausblur=(float*)malloc(gaussize*sizeof(float));
    float norm=0;
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize-1)/2,2)/(2*stdev*stdev));
        norm+=gausblur[i];
    }
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    //second 
    if (p2>0)
    {
        stdev=p2;
    }
    float stdev2=1.2137;
    int gaussize2=(2*int(stdev2)-1)*3;
    if (int(stdev2)==0)
    {
        gaussize2=3;
    }
    float* gausblur2=(float*)malloc(gaussize2*sizeof(float));
    norm=0;
    for (int i=0;i<gaussize2;i++)
    {
        gausblur2[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize2-1)/2,2)/(2*stdev2*stdev2));
        norm+=gausblur2[i];
    }
    for (int i=0;i<gaussize2;i++)
    {
        gausblur2[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    //difference of gaussians
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    sepfilter(r,g,b,rnew,gnew,bnew,width,height,gausblur,gausblur,gaussize);
    sepfilter(r2,g2,b2,rnew,gnew,bnew,width,height,gausblur2,gausblur2,gaussize2);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=r[i*width+j]-r2[i*width+j];
            g[i*width+j]=g[i*width+j]-g2[i*width+j];
            b[i*width+j]=b[i*width+j]-b2[i*width+j];
        }
    }
    free(gausblur); 
    free(gausblur2);
    free(r2);
    free(g2);
    free(b2);
}
break;


case 5: //identity separable
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* identitysep=(float*)malloc(blursize*sizeof(float));
    for (int i=0;i<blursize;i++)
    {
        if (i!=(blursize-1)/2)
        {
            identitysep[i]=0.0;
        }
        else
        {
            identitysep[i]=1.0;
        }
    }
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    sepfilter(r,g,b,rnew,gnew,bnew,width,height,identitysep,identitysep,blursize);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    free(identitysep);

}
break;


case 6: //unsharp mask 3x3 very slo
{
    float unsharp[9]={0,0,0,0,1,0,0,0,0};
    float crs[]={0,1,0,1,1,1,0,1,0};
    float sharp=6;
    if (p1>0)
    {
        sharp=p1;
    }
    float nor=0;
    for(int i=0; i<3; i++)
    {
        for(int j=0; j<3; j++)
        {
            unsharp[i*3+j]+=(sharp*(identity3[i*3+j]-(crs[i*3+j]/sharp)));
            //std::cout<<unsharp[i*3+j]<<" ";
            nor+=unsharp[i*3+j];
        }
    }
    for(int i=0; i<3; i++)
    {
        for(int j=0; j<3; j++)
        {
            unsharp[i*3+j]/=nor;
           //std::cout<<unsharp[i*3+j]<<" ";
        }
    }
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    fblur(r,g,b,rnew,gnew,bnew,width,height,unsharp,3);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
}
break;


case 7:// box blur slo
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* boxblur=(float*)malloc(blursize*blursize*sizeof(float));
    float blooor=float(blursize*blursize);
    for(int i=0;i<blursize;i++)
    {
        for(int j=0;j<blursize;j++)
        {
            boxblur[i*blursize+j]=1.0/blooor;
        }
    }
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    fblur(r,g,b,rnew,gnew,bnew,width,height,boxblur,blursize);
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;    
    free(boxblur);
}
break;
    
default:
{
    std::cout<<"Unknown option. Aboring program.";
    return 0;
}
break;
}
 


/////////IDENTITY TRANSFORMATION SEPARABLE/////////////////////////////////////// 

    //float* identity3=(float*)malloc(9*sizeof(float));
    //memcpy((int**)id3,identity3,9*sizeof(float));

///////BOX BLUR///////////////////////////////////////////////////////////    


        //std::cout<<boxblursep[1]; 
    /*    std::cout<<std::endl;        
    for(int i=0;i<blursize;i++)
    {
        for(int j=0;j<blursize;j++)
        {
            std::cout<<boxblur[i*blursize+j];
        }

    }*/
    //float boxblur[3][3]={{0,0,0},{0,1,0},{0,0,0}};

 //    std::cout<<"aeeeeee"<<std::endl;




    










//CHOOSE WISELY

//fblur(r,g,b,width,height,boxblur,3);




//fblur(r,g,b,width,height,unsharp,3);









/* 



 //whatever the hell this is
    //std::cout<<"aeeeeee"<<std::endl;

    
    /*for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=r[i*width+j]+r2[i*width+j];
            g[i*width+j]=g[i*width+j]+g2[i*width+j];
            b[i*width+j]=b[i*width+j]+b2[i*width+j];
            //std::cout<<int(data[i*width*nchannels+j*nchannels])<<int(data[i*width*nchannels+j*nchannels+1])<<int(data[i*width*nchannels+j*nchannels+2])<<std::endl;
        }
    }  */










//////////////////////////////CLAMP/////////
    clamp(r,g,b,width,height);

//////////OUTPUT/////////////////OUTPUT////////////OUTPUT////////////////////////////////////////OUTPUT////////
    for(unsigned int i=0; i<height; i++)
    {
        for(unsigned int j=0; j<width; j++)
        {
            data[i*width*nchannels+j*nchannels]=(int)round(r[i*width+j]);
            data[i*width*nchannels+j*nchannels+1]=(int)round(g[i*width+j]);
            data[i*width*nchannels+j*nchannels+2]=(int)round(b[i*width+j]);
        }
    } 
    
   /*     for(unsigned int i=0; i<height; i++)
    {
        for(unsigned int j=0; j<width; j++)
        {
            data[i*j*nchannels]=r[i][j];
            data[i*j*nchannels+1]=g[i][j];
            data[i*j*nchannels+2]=b[i][j];
        }
    }  */
    //std::cout<<int(data[2]);
    //std::cout<<std::endl;





///////////////////////////WRITE
    stbi_write_bmp(destination.c_str(),width,height,nchannels, data);


/////////F////////////////////REEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEE


free(r);
free(g);
free(b);
    free(rnew);
    free(gnew);
    free(bnew);

    
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    //t0=wtime()-t0;
    //std::cout<<"time elapsed during the entire program execution meethod old: "<<t0<<" s."<<std::endl;
    std::cout<<"time elapsed during the last part of program: "<<elapsedtime<<" s."<<std::endl;   
    std::cout<<"time elapsed during the entire program execution: "<<totaltime<<" s."<<std::endl;
    return 0;
}
