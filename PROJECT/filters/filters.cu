#include "hip/hip_runtime.h"
#ifndef _POSIX_C_SOURCE
#define _POSIX_C_SOURCE 199309L
#endif
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <unistd.h>


#define STB_IMAGE_IMPLEMENTATION
#include "../Common/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Common/stb_image_write.h"








double wtime()
{
static int sec = -1;
struct timespec tv;
clock_gettime(CLOCK_MONOTONIC, &tv);
if (sec < 0) sec = tv.tv_sec;
return (tv.tv_sec - sec) + 1.0e-9*tv.tv_nsec;
}



__global__ void fblur(float* r, float* g, float* b, float* r2, float* g2, float* b2, int width, int height, float*  blur, int blursize) //standard blur
{   
    __syncthreads();
    long long int idx= threadIdx.x + blockIdx.x * blockDim.x;
    int j=int(idx%width);
    int i=int(idx/width);

    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;

    int shift=(blursize-1)/2;
__syncthreads();
    //MAIN BODY
    if(i>=shift && i<height-shift && j>=shift && j<width-shift)
    {

            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(i-shift+k)*width + (j-shift+l)]);
                    green_tmp+=blur[k*blursize+l]*(g[(i-shift+k)*width + (j-shift+l)]);
                    blu_tmp+=blur[k*blursize+l]*(b[(i-shift+k)*width + (j-shift+l)]);
                    //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
                }
                
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    //EDGE HANDLING HERE IS BROKEN SO IT HAS BEEN TURNED OFF
    //EDG TOP
    /*__syncthreads();
    if(i>=0 && i<shift && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDG left
    if(i>=0 && i<height && j>=0 && j<shift)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDGE DOWN
    if(i>=height-shift && i<height && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    if(i>=0 && i<height && j>=width-shift && j<width)//EDGE RIGHT
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }  */ 
        //PASS TO rgb
    if(i>=0&& i<height && j>=0 && j<=width)
    {
            r[i*width+j]=r2[i*width+j];
            g[i*width+j]=g2[i*width+j];
            b[i*width+j]=b2[i*width+j];
    }
}

/* __global__ void fedge(float* r, float* g, float* b, float* r2, float* g2, float* b2, int width, int height, float*  blur, int blursize) //poor attempt at trying to gain edge filtering back
{
    long long int idx= threadIdx.x + blockIdx.x * blockDim.x;
    int j=int(idx%width);
    int i=int(idx/width);

    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;

    int shift=(blursize-1)/2;
    //EDG TOP
    __syncthreads();
    if(i>=0 && i<shift && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDG left
    if(i>=0 && i<height && j>=0 && j<shift)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*r[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    green_tmp+=blur[k*blursize+l]*g[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    blu_tmp+=blur[k*blursize+l]*b[abs(((i-shift)+k))*width + abs((j-shift+l))];
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDGE DOWN
    if(i>=height-shift && i<height && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    if(i>=0 && i<height && j>=width-shift && j<width)//EDGE RIGHT
    {
            for (int k=0; k<blursize; k++)
            {
                for (int l=0; l<blursize; l++)
                {
                    red_tmp+=blur[k*blursize+l]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    green_tmp+=blur[k*blursize+l]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    blu_tmp+=blur[k*blursize+l]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + width-1-abs(width-1-(j-shift+l))]);
                    
                }
            }
            r2[i*width+j]=(red_tmp);
            g2[i*width+j]=(green_tmp);
            b2[i*width+j]=(blu_tmp);
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    } 
    
    __syncthreads();
    //PASS TO rgb
    if(i<shift || i>=height-shift || j<shift || j>=width-shift)
    { 
            r[i*width+j]=r2[i*width+j];
            g[i*width+j]=g2[i*width+j];
            b[i*width+j]=b2[i*width+j];
    }
}*/


__global__ void sepfilterH(float* r, float* g, float* b, float* rnew, float* gnew, float* bnew, int width, int height, float*  blurh, int blursize)
{   __syncthreads();
    long long int idx= threadIdx.x + blockIdx.x * blockDim.x;
    int j=int(idx%width);
    int i=int(idx/width);
    
    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;
    __syncthreads();
    int shift=(blursize-1)/2;
    //MAIN BLUR HORIZONTAL, NO LEFT & RIGHT EDGES
    if(i>=0 && i<height && j>=shift && j<(width-shift))
    {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + j-shift+l]);
                green_tmp+=blurh[l]*(gnew[i*width + j-shift+l]);
                blu_tmp+=blurh[l]*(bnew[i*width + j-shift+l]);
          
                //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
            }
       
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;

            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDGE LEFT HORIZONTAL PART
    //VERTICAL PART WAS DONE IN MAIN BLUR VERTICAL
    if(i>=0 && i<height && j>=0 && j<shift)
    {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + abs((j-shift+l))]);
                green_tmp+=blurh[l]*(gnew[i*width + abs((j-shift+l))]);
                blu_tmp+=blurh[l]*(bnew[i*width + abs((j-shift+l))]);
            }
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDG RIGHT
    if(i>=0 && i<height && j>=(width-shift) && j<width)
    {
            for (int l=0; l<blursize; l++)
            {
                red_tmp+=blurh[l]*(rnew[i*width + width-1-abs(width-1-(j-shift+l))]);
                green_tmp+=blurh[l]*(gnew[i*width + width-1-abs(width-1-(j-shift+l))]);
                blu_tmp+=blurh[l]*(bnew[i*width + width-1-abs(width-1-(j-shift+l))]);
            }
            r[i*width+j]=red_tmp;
            g[i*width+j]=green_tmp;
            b[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }

}


__global__ void sepfilterV(float* r, float* g, float* b, float* rnew, float* gnew, float* bnew, int width, int height, float* blurv, int blursize) //blur function for spearable filters, much speed wow
{
    __syncthreads();
    long long int idx= threadIdx.x + blockIdx.x * blockDim.x;
    int j=int(idx%width);
    int i=int(idx/width);
    
    //int i=blockIdx.x*blockDim.x+threadIdx.x;
    //int j=blockIdx.y*blockDim.y+threadIdx.y; 

    float red_tmp=0;
    float green_tmp=0;
    float blu_tmp=0;

    int shift=(blursize-1)/2;
    __syncthreads();
    //MAIN BLUR VERTICAL 
    if(i>=shift && i<(height-shift) && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {

                //red_tmp+=blurh[k]*float(r[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //green_tmp+=blurh[k]*float(g[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //blu_tmp+=blurh[k]*float(b[(i-((blursize-1)/2)+k)*width + (j-((blursize-1)/2))]);
                //blu_tmp+=blur[k*blursize+l]*float(b[i-(blursize-1)/2+k][j-(blursize-1)/2+l]);
          
                red_tmp+=blurv[k]*(r[(i-shift)*width+k*width + j]);
                green_tmp+=blurv[k]*(g[(i-shift)*width+k*width + j]);
                blu_tmp+=blurv[k]*(b[(i-shift)*width+k*width + j]);
      
                
            }

            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;

            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;

    }
 
    __syncthreads();
    //EDGE UP USING MIRROR & VERTICAL BLUR
    //SOMEBODY ONCE TOLD ME THAT IF STATEMENTS ARE SLOW
    if(i>=0 && i<shift && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                red_tmp+=blurv[k]*(r[abs(((i-shift)+k))*width + j]);
                green_tmp+=blurv[k]*(g[abs(((i-shift)+k))*width + j]);
                blu_tmp+=blurv[k]*(b[abs(((i-shift)+k))*width + j]);
            }
            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    __syncthreads();
    //EDGE DOWN
    if(i>=(height-shift) && i<height && j>=0 && j<width)
    {
            for (int k=0; k<blursize; k++)
            {
                red_tmp+=blurv[k]*(r[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
                green_tmp+=blurv[k]*(g[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
                blu_tmp+=blurv[k]*(b[(-1+height-abs(height-1-(i-shift+k)))*width + j]);
            }
            rnew[i*width+j]=red_tmp;
            gnew[i*width+j]=green_tmp;
            bnew[i*width+j]=blu_tmp;
            red_tmp=0;
            green_tmp=0;
            blu_tmp=0;
    }
    

}



void clamp(float* r, float* g, float* b, int width, int height) /////CLAMP TO RGB RANGE
{
    for(unsigned int i=0; i<height; i++)
    {
        for(unsigned int j=0; j<width; j++)
        {
            if (r[i*width+j]>255.0)
            {
                r[i*width+j]=255.0;
            } 
            else if (r[i*width+j]<0)
            {
                r[i*width+j]=0;
            }
            
            if (g[i*width+j]>255.0)
            {
                g[i*width+j]=255.0;
            }
            else if (g[i*width+j]<0)
            {
                g[i*width+j]=0;
            }
            
            
            if (b[i*width+j]>255.0)
            {
                b[i*width+j]=255.0;
            }
            else if (b[i*width+j]<0)
            {
                b[i*width+j]=0;
            }
            
            
        }
    }
}





int main()
{
    int deviceId;
    hipGetDevice(&deviceId);
    std::cout<<"welcome. input (separated by spaces) your source image path, your chosen destination path and desired operation and parameters for it."<<std::endl<<"0: gaussian blur (parameter: standard deviation (px, float))  ///  1: box blur (parameter: size (px, int, odd))"<<std::endl<<"2: circular box blur (bokeh) (parameter: diameter (px, int, odd))  ///  3: sobel edge detection (parameters: vertical detection, horizontal detection (bool))"<<std::endl<<"4: difference of gaussians edge detection (2 parameters: stdev1, stdev2 (float>0))  ///  5:identity transformation (parameter: size (int)) (useless)"<<std::endl<<"6: unsharp mask simple (parameter: strength (float))  ///  7: unseparated (slow) box blur (parameter: size(px, int, odd))"<<std::endl<<"always type in two numbers as parameters, if the process uses only one parameter set the second one to whatever. 0 or less sets parameter to its default value"<<std::endl;
    std::cout<<"example input: '../samples/default.bmp ../output/test.bmp 0 7 0'"<<std::endl<<std::endl;
    std::string source="default.bmp";
    std::string destination="output/test.bmp";
    int width=0;
    int height=0;
    int sw=0;
    float p1=0;
    float p2=0;
    int p3=0;
    //dim3 threadsperblock(8,8,1);
    dim3 threadsperblock(64,1,1);
    int nchannels=3;
    std::cin>>source>>destination>>sw>>p1>>p2;
    double totaltime=0;
    double elapsedtime=wtime();
    double t0=elapsedtime;
    //check if image exists, is valid format and if yes then query values of width, height and channels
    if (stbi_info(source.c_str(), &width, &height, &nchannels)==0)
    {
        std::cout<<"Invalid source. Aborting mission.";
        return 0;
    }

    if(((long long int)(width)*height)>(2147482137*(threadsperblock.x*threadsperblock.x)))//check if not too big for not using grid stride loops. should be okay for up to many many megapixels to a point where its sligthly sus if you have such a big image
    {
        std::cout<<"listen to me carefully. Tomorrow at 9:37 you have a plane to mexico. I will email you the ticket right away. After you walk out of the airport below a red phone booth is a small container, open it using the secret password: 'hajduszoboszlo'. Therein you will find your new ID card, 3000 pesos and keys to an apartment on the opposite side. From now on your name is Juan Pablo Fernandez Maria FC Barcelona Yanush Sergio Vasilii Shevchenko and you are a russian immigrant from Romania. You work in a barber shop 2 km from the airport. Good luck. Forget about your previous life and maintain a low profile, ditch all of your connections, even with the nvidia customer service."<<std::endl;
        return 0;
    }
    
    //dim3 numberofblocks(((width-1) / threadsperblock.x) + 1, ((height-1) / threadsperblock.y) + 1, 1);
    dim3 numberofblocks((((width*height)-1) / threadsperblock.x) + 1, 1, 1);

    unsigned char *data=stbi_load(source.c_str(), &width, &height, &nchannels, 0);
    //std::cout<<wtime()<<std::endl;    


    float* r;
    float* g;
    float* b;
    float* rnew;
    float* gnew;
    float* bnew; 
    hipMallocManaged(&r, width*height*sizeof(float));
    hipMallocManaged(&g, width*height*sizeof(float));
    hipMallocManaged(&b, width*height*sizeof(float));
    hipMallocManaged(&rnew, width*height*sizeof(float));
    hipMallocManaged(&gnew, width*height*sizeof(float));
    hipMallocManaged(&bnew, width*height*sizeof(float));
    hipMemPrefetchAsync(r, width*height*sizeof(float), deviceId);
    hipMemPrefetchAsync(g, width*height*sizeof(float), deviceId);
    hipMemPrefetchAsync(b, width*height*sizeof(float), deviceId);
    hipMemPrefetchAsync(rnew, width*height*sizeof(float), deviceId);
    hipMemPrefetchAsync(gnew, width*height*sizeof(float), deviceId);
    hipMemPrefetchAsync(bnew, width*height*sizeof(float), deviceId);
   


    //std::cout<<"aeeeeee"<<r[0]<<std::endl;

    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=float(data[i*width*nchannels+j*nchannels]);
            g[i*width+j]=float(data[i*width*nchannels+j*nchannels+1]);
            b[i*width+j]=float(data[i*width*nchannels+j*nchannels+2]);
            //std::cout<<int(data[i*width*nchannels+j*nchannels])<<int(data[i*width*nchannels+j*nchannels+1])<<int(data[i*width*nchannels+j*nchannels+2])<<std::endl;
        }
    }  

    //std::cout<<"aeeeeee"<<std::endl;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////IMYDŻ LOADED
 
    int blursize=3;//keep this odd
    float identity3[]={0,0,0,0,1,0,0,0,0};

switch (sw)
{
case 0:// gausjan
{    ////////GAUS BLUR/////////////    
    float stdev=2;//default
    if (p1>0)
    {
        stdev=p1;
    }
    int gaussize=(2*int(stdev)-1)*3;// 3 sigma wide blur
    if (int(stdev)==0)
    {
        gaussize=3;
    }
    float* gausblur;
    hipMallocManaged(&gausblur, gaussize*sizeof(float));
    float norm=0;
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize-1)/2,2)/(2*stdev*stdev));
        norm+=gausblur[i];
    }
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    hipMemPrefetchAsync(gausblur, gaussize*sizeof(float), deviceId);  
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    sepfilterV<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gausblur,gaussize);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gausblur,gaussize);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed during filtering: "<<elapsedtime<<" s."<<std::endl;
    hipFree(gausblur);
}
break;
    
  
case 1://box blur (separable)
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* boxblursep;
    hipMallocManaged(&boxblursep, blursize*sizeof(float));
    for(int i=0;i<blursize;i++)
    {
        boxblursep[i]=1.0/float(blursize);
        //std::cout<<boxblursep[i]<<", ";
    }
    hipMemPrefetchAsync(boxblursep, blursize*sizeof(float), deviceId);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl; 
    sepfilterV<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,boxblursep,blursize);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,boxblursep,blursize);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    hipFree(boxblursep);
}
break;


  ///////BOKEH (CIRCULAR BOX BLUR) obviously one might say there are no perfect circles in any image composed of pixels and while this is one hundred percent true one may see that they cant actually see the uncircleness beyond a certain point which I shall now take the advantage of
case 2://bokeh very slow not separed
{
    int diameter=5;/////diameter in pixels keep this odd pls
    if (p1>0)
    {
        diameter=(int)(2*ceil(p1/2)-1);
    }
    int radius=(diameter-1)/2;
    float* bokeh;
    hipMallocManaged(&bokeh, diameter*diameter*sizeof(float));
    float R=0;
    int sb=0;
    for(int i=0; i<diameter; i++)//initialise
    {
        for(int j=0; j<diameter; j++)
        {
            R=sqrt((j-radius)*(j-radius)+(i-radius)*(i-radius));
            if (R<=radius)
            {
                bokeh[i*diameter+j]=1;
                sb+=1;
            }
            else
            {
                bokeh[i*diameter+j]=0;
            }
        }
    }
    for(int i=0; i<diameter; i++)
    {
        for(int j=0; j<diameter; j++)
        {
            bokeh[i*diameter+j]/=float(sb);
            //std::cout<<bokeh[i*diameter+j]<<" ";
        }
    }
    hipMemPrefetchAsync(bokeh, diameter*diameter*sizeof(float), deviceId);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;  
    fblur<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,bokeh,diameter);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    hipFree(bokeh);
}
break;

case 3://sobel
{
    bool ver=(bool)int(p1);
    bool hor=(bool)int(p2);
    float* r2;
    float* g2;
    float* b2;
    hipMallocManaged(&r2,width*height*sizeof(float));
    hipMallocManaged(&g2,width*height*sizeof(float));
    hipMallocManaged(&b2,width*height*sizeof(float));
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r2[i*width+j]=r[i*width+j];
            g2[i*width+j]=g[i*width+j];
            b2[i*width+j]=b[i*width+j];
        }
    }  
    float gxv[]={1,2,1};
    float gxh[]={1,0,-1};
    float gyv[]={1,0,-1};
    float gyh[]={1,2,1};
    hipMemPrefetchAsync(&r2,width*height*sizeof(float),deviceId);
    hipMemPrefetchAsync(&g2,width*height*sizeof(float),deviceId);
    hipMemPrefetchAsync(&b2,width*height*sizeof(float),deviceId);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;   
    sepfilterV<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gxv,3);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gxh,3);
    hipDeviceSynchronize();
    sepfilterV<<<numberofblocks,threadsperblock>>>(r2,g2,b2,rnew,gnew,bnew,width,height,gyv,3);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r2,g2,b2,rnew,gnew,bnew,width,height,gyh,3);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    
    if(ver && hor)
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=sqrt(r[i*width+j]*r[i*width+j]+r2[i*width+j]*r2[i*width+j]);
                g[i*width+j]=sqrt(g[i*width+j]*g[i*width+j]+g2[i*width+j]*g2[i*width+j]);
                b[i*width+j]=sqrt(b[i*width+j]*b[i*width+j]+b2[i*width+j]*b2[i*width+j]);
            }
        }  
    }
    else if (ver)
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=r2[i*width+j];
                g[i*width+j]=g2[i*width+j];
                b[i*width+j]=b2[i*width+j];
            }
        }
    }
    else if (hor)
    {
        
    }
    else
    {
        for(int i=0; i<height; i++)//combine vertical and horizontal
        {
            for(int j=0; j<width; j++)
            {
                r[i*width+j]=0;
                g[i*width+j]=0;
                b[i*width+j]=0;
            }
        }
    }
    hipFree(r2);
    hipFree(g2);
    hipFree(b2);
}
break;


case 4: //TWO GAUS
{
    float* r2;
    float* g2;
    float* b2;
    float* gausblur;
    float* gausblur2;
    hipMallocManaged(&r2,width*height*sizeof(float));
    hipMallocManaged(&g2,width*height*sizeof(float));
    hipMallocManaged(&b2,width*height*sizeof(float));

    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r2[i*width+j]=r[i*width+j];
            g2[i*width+j]=g[i*width+j];
            b2[i*width+j]=b[i*width+j];
        }
    }  
    hipMemPrefetchAsync(&r2,width*height*sizeof(float),deviceId);
    hipMemPrefetchAsync(&g2,width*height*sizeof(float),deviceId);
    hipMemPrefetchAsync(&b2,width*height*sizeof(float),deviceId);
    float stdev=2;//default
    if (p1>0)
    {
        stdev=p1;
    }
    int gaussize=(2*int(stdev)-1)*3;// 3 sigma wide blur
    if (int(stdev)==0)
    {
        gaussize=3;
    }
    hipMallocManaged(&gausblur,gaussize*sizeof(float));
    float norm=0;
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize-1)/2,2)/(2*stdev*stdev));
        norm+=gausblur[i];
    }
    for (int i=0;i<gaussize;i++)
    {
        gausblur[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    hipMemPrefetchAsync(&gausblur,gaussize*sizeof(float),deviceId);
    //second 
    if (p2>0)
    {
        stdev=p2;
    }
    float stdev2=1.2137;
    int gaussize2=(2*int(stdev2)-1)*3;
    if (int(stdev2)==0)
    {
        gaussize2=3;
    }
    hipMallocManaged(&gausblur2,gaussize2*sizeof(float));
    norm=0;
    for (int i=0;i<gaussize2;i++)
    {
        gausblur2[i]=1/sqrt(2*3.14159)*exp(-(float)pow(i-(gaussize2-1)/2,2)/(2*stdev2*stdev2));
        norm+=gausblur2[i];
    }
    for (int i=0;i<gaussize2;i++)
    {
        gausblur2[i]/=norm;
        //std::cout<<gausblur[i]<<' ';
    }
    hipMemPrefetchAsync(&gausblur2,gaussize2*sizeof(float),deviceId);

    //difference of gaussians
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    hipDeviceSynchronize();
    sepfilterV<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gausblur,gaussize);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,gausblur,gaussize);
    hipDeviceSynchronize();
    sepfilterV<<<numberofblocks,threadsperblock>>>(r2,g2,b2,rnew,gnew,bnew,width,height,gausblur2,gaussize2);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r2,g2,b2,rnew,gnew,bnew,width,height,gausblur2,gaussize2);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=r[i*width+j]-r2[i*width+j];
            g[i*width+j]=g[i*width+j]-g2[i*width+j];
            b[i*width+j]=b[i*width+j]-b2[i*width+j];
        }
    }
    hipFree(gausblur); 
    hipFree(gausblur2);
    hipFree(r2);
    hipFree(g2);
    hipFree(b2);
}
break;


case 5: //identity separable
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* identitysep;
    hipMallocManaged(&identitysep,blursize*sizeof(float));
    for (int i=0;i<blursize;i++)
    {
        if (i!=(blursize-1)/2)
        {
            identitysep[i]=0.0;
        }
        else
        {
            identitysep[i]=1.0;
        }
    }
    hipMemPrefetchAsync(&identitysep,blursize*sizeof(float),deviceId);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    hipDeviceSynchronize();
    sepfilterV<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,identitysep,blursize);
    hipDeviceSynchronize();
    sepfilterH<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,identitysep,blursize);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
    free(identitysep);

}
break;


case 6: //unsharp mask 3x3 very slow
{
    float unsharp[9]={0,0,0,0,1,0,0,0,0};
    float crs[]={0,1,0,1,1,1,0,1,0};
    float sharp=6;
    if (p1>0)
    {
        sharp=p1;
    }
    float nor=0;
    for(int i=0; i<3; i++)
    {
        for(int j=0; j<3; j++)
        {
            unsharp[i*3+j]+=(sharp*(identity3[i*3+j]-(crs[i*3+j]/sharp)));
            //std::cout<<unsharp[i*3+j]<<" ";
            nor+=unsharp[i*3+j];
        }
    }
    for(int i=0; i<3; i++)
    {
        for(int j=0; j<3; j++)
        {
            unsharp[i*3+j]/=nor;
           //std::cout<<unsharp[i*3+j]<<" ";
        }
    }
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    fblur<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,unsharp,3);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;
}
break;


case 7:// box blur slo
{
    if (p1>0)
    {
        blursize=(int)(2*ceil(p1/2)-1);
    }
    float* boxblur;
    hipMallocManaged(&boxblur, blursize*blursize*sizeof(float));
    float blooor=float(blursize*blursize);
    for(int i=0;i<blursize;i++)
    {
        for(int j=0;j<blursize;j++)
        {
            boxblur[i*blursize+j]=1.0/blooor;
        }
    }
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed from starting program to beginning of filtering: "<<elapsedtime<<" s."<<std::endl;
    fblur<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,boxblur,blursize);
    //hipDeviceSynchronize();
    //fedge<<<numberofblocks,threadsperblock>>>(r,g,b,rnew,gnew,bnew,width,height,boxblur,blursize);
    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    std::cout<<"time elapsed during  filtering: "<<elapsedtime<<" s."<<std::endl;    
    hipFree(boxblur);
}
break;
    
default:
{
    std::cout<<"Unknown option. Aboring program.";
    return 0;
}
break;

}


    

/////////IDENTITY TRANSFORMATION SEPARABLE/////////////////////////////////////// 

    //float* identity3=(float*)malloc(9*sizeof(float));
    //memcpy((int**)id3,identity3,9*sizeof(float));

///////BOX BLUR///////////////////////////////////////////////////////////    


        //std::cout<<boxblursep[1]; 
    /*    std::cout<<std::endl;        
    for(int i=0;i<blursize;i++)
    {
        for(int j=0;j<blursize;j++)
        {
            std::cout<<boxblur[i*blursize+j];
        }

    }*/
    //float boxblur[3][3]={{0,0,0},{0,1,0},{0,0,0}};

 //    std::cout<<"aeeeeee"<<std::endl;




    










//CHOOSE WISELY

//fblur(r,g,b,width,height,boxblur,3);




//fblur(r,g,b,width,height,unsharp,3);









/* 



 //whatever the hell this is
    //std::cout<<"aeeeeee"<<std::endl;

    
    /*for(int i=0; i<height; i++)
    {
        for(int j=0; j<width; j++)
        {
            r[i*width+j]=r[i*width+j]+r2[i*width+j];
            g[i*width+j]=g[i*width+j]+g2[i*width+j];
            b[i*width+j]=b[i*width+j]+b2[i*width+j];
            //std::cout<<int(data[i*width*nchannels+j*nchannels])<<int(data[i*width*nchannels+j*nchannels+1])<<int(data[i*width*nchannels+j*nchannels+2])<<std::endl;
        }
    }  */





    hipMemPrefetchAsync(r, width*height*sizeof(float), hipCpuDeviceId);
    hipMemPrefetchAsync(g, width*height*sizeof(float), hipCpuDeviceId);
    hipMemPrefetchAsync(b, width*height*sizeof(float), hipCpuDeviceId);


hipDeviceSynchronize();

//////////////////////////////CLAMP/////////
    clamp(r,g,b,width,height);

//////////OUTPUT/////////////////OUTPUT////////////OUTPUT////////////////////////////////////////OUTPUT////////
    for(unsigned int i=0; i<height; i++)
    {
        for(unsigned int j=0; j<width; j++)
        {
            data[i*width*nchannels+j*nchannels]=(int)round(r[i*width+j]);
            data[i*width*nchannels+j*nchannels+1]=(int)round(g[i*width+j]);
            data[i*width*nchannels+j*nchannels+2]=(int)round(b[i*width+j]);
        }
    } 
    
   /*     for(unsigned int i=0; i<height; i++)
    {
        for(unsigned int j=0; j<width; j++)
        {
            data[i*j*nchannels]=r[i][j];
            data[i*j*nchannels+1]=g[i][j];
            data[i*j*nchannels+2]=b[i][j];
        }
    }  */
    //std::cout<<int(data[2]);
    //std::cout<<std::endl;





///////////////////////////WRITE
    stbi_write_bmp(destination.c_str(),width,height,nchannels, data);


/////////F////////////////////REEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEEE

hipFree(rnew);
hipFree(gnew);
hipFree(bnew); 

hipFree(r);
hipFree(g);
hipFree(b);



    hipDeviceSynchronize();
    elapsedtime=wtime()-elapsedtime;
    totaltime+=elapsedtime;
    //t0=wtime()-t0;
    //std::cout<<"time elapsed during the entire program execution meethod old: "<<t0<<" s."<<std::endl;
    std::cout<<"time elapsed during the last part of program: "<<elapsedtime<<" s."<<std::endl;  
    std::cout<<"time elapsed during the entire program execution: "<<totaltime<<" s."<<std::endl;
    return 0;
}